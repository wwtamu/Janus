#include "hip/hip_runtime.h"
#include "utility.cuh"

static __global__ void encapsulate_kernel(int width, int height, int* artifact, int2* characters, int max)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if (x < width && y < height)
	{
		if (artifact[y*width + x] == 0) {

			int2 character[4] = {
				{ x, y },
				{ x, y },
				{ x, y },
				{ x, y }
			};

			if (x != 0 && y != 0) {
				character[0].x--; 
				character[0].y--;
				character[1].y--;
				character[3].x--;
			}
			else if (x != width && y != height) {
				character[1].x++;
				character[2].x++;
				character[2].y++;
				character[3].y++;
			}

			bool finished = false, restart = false;

			int count = 0;

			while (!finished && count <= max) {

				if (!restart) {
					for (int i = 0; i < abs(character[0].x - character[1].x); i++) {

						if (artifact[character[0].y*width + (character[0].x + i)] == 0) {
							restart = true;
							character[0].y--;
							character[1].y--;

							break;
						}
					}
				}

				if (!restart) {
					for (int i = 0; i < abs(character[1].y - character[2].y); i++) {

						if (artifact[(character[1].y + i)*width + character[1].x] == 0) {
							restart = true;
							character[1].x++;
							character[2].x++;

							break;
						}
					}
				}

				if (!restart) {
					for (int i = 0; i < abs(character[2].x - character[3].x); i++) {

						if (artifact[character[2].y*width + (character[2].x - i)] == 0) {
							restart = true;
							character[2].y++;
							character[3].y++;

							break;
						}
					}
				}

				if (!restart) {
					for (int i = 0; i < abs(character[3].y - character[0].y); i++) {

						if (artifact[(character[3].y - i)*width + character[3].x] == 0) {
							restart = true;
							character[3].x--;
							character[0].x--;

							break;
						}
					}
				}

				if (!restart) {
					finished = true;
				}
				else {
					restart = false;
				}

				count++;

			}

			if (count < max) {

				//V[z][y][x] : z*s(y)*s(x) + y*s(x) + x

				characters[x*height * 4 + y * 4 + 0] = { character[0].x, character[0].y };
				characters[x*height * 4 + y * 4 + 1] = { character[1].x, character[1].y };
				characters[x*height * 4 + y * 4 + 2] = { character[2].x, character[2].y };
				characters[x*height * 4 + y * 4 + 3] = { character[3].x, character[3].y };
				
				for (int i = 0; i < abs(character[0].x - character[1].x); i++) {
					artifact[character[0].y*width + (character[0].x + i)] = 150;
				}
				for (int i = 0; i < abs(character[1].y - character[2].y); i++) {
					artifact[(character[1].y + i)*width + character[1].x] = 150;
				}
				for (int i = 0; i < abs(character[2].x - character[3].x); i++) {
					artifact[character[2].y*width + (character[2].x - i)] = 150;
				}
				for (int i = 0; i < abs(character[3].y - character[0].y); i++) {
					artifact[(character[3].y - i)*width + character[3].x] = 150;
				}
				
			}

		}

	}

}

static hipError_t encapsulate(int width, int height, int* artifact, int2* characters, int max, int gx, int gy, int bx, int by)
{
	hipError_t cudaStatusA;
	hipError_t cudaStatusB;
	{
		dim3 grid(gx, gy, 1);
		dim3 block(bx, by, 1);
		encapsulate_kernel<<<grid, block>>>(width, height, artifact, characters, max);
	}
	cudaStatusA = hipGetLastError(); if (cudaStatusA != hipSuccess) { fprintf(stderr, "encapsulate_kernel launch failed: %s\n", hipGetErrorString(cudaStatusA)); goto Error; }
	cudaStatusB = hipDeviceSynchronize(); if (cudaStatusB != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching encapsulate_kernel!\n", cudaStatusB); goto Error; }
	;
Error:
	;
	return (cudaStatusA != hipSuccess) ? cudaStatusA : (cudaStatusB != hipSuccess) ? cudaStatusB : hipSuccess;
}

Mat preprocess(Mat image, int threshold) {

	chrono::high_resolution_clock::time_point start = chrono::high_resolution_clock::now();

	cv::cvtColor(image, image, CV_BGR2GRAY);
	cv::threshold(image, image, threshold, 255, CV_THRESH_BINARY);

	chrono::high_resolution_clock::time_point stop = chrono::high_resolution_clock::now();

	cout << "preprocess: " << chrono::duration_cast<chrono::milliseconds>(stop - start).count() << " milliseconds" << endl << endl;

	if (DISPLAY_AFTER_PREPROCESS) {

		namedWindow("Janus", WINDOW_AUTOSIZE);
		imshow("Janus", image);

		waitKey(0);

	}

	return image;
}

vector<character> extract(Mat image, int max, int sized, int quadrants, string filename) {

	int GRID_X = ((image.cols + THREAD_X - 1) / THREAD_X);
	int GRID_Y = ((image.rows + THREAD_Y - 1) / THREAD_Y);

	chrono::high_resolution_clock::time_point start = chrono::high_resolution_clock::now();

	int* artifact = (int*)malloc(image.cols*image.rows*sizeof(int));

	int2* characters = (int2*)malloc(image.cols*image.rows * 4 * sizeof(int2));


	int* devArtifact; hipMalloc((void **)&devArtifact, image.cols*image.rows*sizeof(int));

	int2* devCharacters; hipMalloc((void **)&devCharacters, image.cols*image.rows * 4 * sizeof(int2));


	for (int x = 0; x < image.cols; x++) {

		for (int y = 0; y < image.rows; y++) {

			//V[z][y][x] : z*s(y)*s(x) + y*s(x) + x

			characters[x*image.rows * 4 + y * 4 + 0] = { -1, -1 };
			characters[x*image.rows * 4 + y * 4 + 1] = { -1, -1 };
			characters[x*image.rows * 4 + y * 4 + 2] = { -1, -1 };
			characters[x*image.rows * 4 + y * 4 + 3] = { -1, -1 };
			
			Scalar intensity = image.at<uchar>(y, x);

			artifact[y*image.cols + x] = (int)intensity.val[0];
		}
	}


	hipMemcpy(devArtifact, artifact, image.cols*image.rows * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(devCharacters, characters, image.cols*image.rows * 4 * sizeof(int2), hipMemcpyHostToDevice);
	

	if (encapsulate(image.cols, image.rows, devArtifact, devCharacters, max, GRID_X, GRID_Y, THREAD_X, THREAD_Y) != hipSuccess) {
		_exit("encapsulation failed", EXIT_FAILURE);
	}


	hipMemcpy(artifact, devArtifact, image.cols*image.rows*sizeof(int), hipMemcpyDeviceToHost);

	hipMemcpy(characters, devCharacters, image.cols*image.rows * 4 * sizeof(int2), hipMemcpyDeviceToHost);

	hipFree(devArtifact);
	hipFree(devCharacters);


	set<box> boxes;

	for (int i = 0; i < image.cols*image.rows * 4; i += 4) {

		if (characters[i].x > 0) {

			if (abs(characters[i + 1].x - characters[i].x) > 2 && abs(characters[i + 2].y - characters[i + 1].y) > 2) {
				boxes.insert(box(make_int2(characters[i].x, characters[i].y), abs(characters[i + 1].x - characters[i].x)/* + 1*/, abs(characters[i + 2].y - characters[i + 1].y)/* + 1*/));
			}

		}

	}

	
	Size size(sized, sized);

	vector<character> tableau;

	set<box>::iterator it;

	int count = 1;

	for (it = boxes.begin(); it != boxes.end(); ++it) {

		box boxed_character = *it;
		
		// how often does it go out of bounds? how far out of bounds does it go? should be handled when making boxed_character
		if ((boxed_character.getLocation().x + boxed_character.getWidth()) <= image.cols && 
			(boxed_character.getLocation().y + boxed_character.getHeight()) <= image.rows) 
		{

			Mat sub_image(image, Rect(boxed_character.getLocation().x, boxed_character.getLocation().y, boxed_character.getWidth(), boxed_character.getHeight()));

			int2 original_size = { sub_image.cols, sub_image.rows };

			if (SHOW_LOCATION) {
				cout << "location: " << boxed_character.getLocation().x << "," << boxed_character.getLocation().y << endl;
			}
			if (SHOW_ORIGINAL_SIZE) {
				cout << "original size: " << original_size.x << "," << original_size.y << endl;
			}

			Mat sized_character;

			resize(sub_image, sized_character, size);

			char* vertical_celled_projection = (char*)malloc(sized_character.cols*sized_character.rows*sizeof(char));
			char* horizontal_celled_projection = (char*)malloc(sized_character.cols*sized_character.rows*sizeof(char));


			char* binary = (char*)malloc(sized_character.cols*sized_character.rows*sizeof(char));

			double* density = (double*)malloc(quadrants*quadrants*sizeof(double));

			int* ones = (int*)malloc(quadrants*quadrants*sizeof(int));
			int* zeroes = (int*)malloc(quadrants*quadrants*sizeof(int));

			int quad_size = sized / quadrants;

			int index = 0;
			int _ones = 0;
			int _zeroes = 0;

			for (int qx = 0; qx < sized_character.cols; qx += quad_size) {
				for (int qy = 0; qy < sized_character.rows; qy += quad_size) {
					for (int x = qx; x < qx + quad_size; x++) {
						for (int y = qy; y < qy + quad_size; y++) {
							if (sized_character.at<uchar>(x, y) == 255) {
								binary[x*sized_character.rows + y] = '0';
								_zeroes++;
							}
							else {
								binary[x*sized_character.rows + y] = '1';
								_ones++;
							}
							vertical_celled_projection[x*sized_character.rows + y] = '0';
							horizontal_celled_projection[x*sized_character.rows + y] = '0';
						}
					}
					ones[index] = _ones;
					zeroes[index] = _zeroes;
					_ones = 0;
					_zeroes = 0;
					index++;
				}

			}
			
			if (SHOW_BINARY_REPRESENTATION) {
				cout << "binary representation" << endl << endl;
				for (int x = 0; x < sized_character.cols; x++) {
					for (int y = 0; y < sized_character.rows; y++) {
						cout << binary[x*sized_character.rows + y];
					}
					cout << endl;
				}
				cout << endl;
			}


			if (SHOW_VERTICAL_CELLED_PROJECTION) {
				cout << "density matrix" << endl << endl;
				for (int x = 0; x < quadrants; x++) {
					for (int y = 0; y < quadrants; y++) {
						density[x*quadrants + y] = (double)ones[x*quadrants + y] / (double)(ones[x*quadrants + y] + zeroes[x*quadrants + y]);
						if (y < quadrants - 1) {
							printf("%.2f,", density[x*quadrants + y]);
						}
						else {
							printf("%.2f", density[x*quadrants + y]);
						}
					}
					cout << endl;
				}
				cout << endl;
			}

			int slice_gap = sized / quadrants;

			for (int sx = 0; sx < sized_character.cols; sx += slice_gap) {
				for (int x = sx; x < sx + slice_gap; x++) {
					for (int y = 0; y < sized_character.rows; y++) {
						if (binary[x*sized_character.rows + y] == '1') {
							vertical_celled_projection[sx*sized_character.rows + y] = '1';
						}
					}
				}
			}

			for (int sy = 0; sy < sized_character.rows; sy += slice_gap) {
				for (int y = sy; y < sy + slice_gap; y++) {
					for (int x = 0; x < sized_character.cols; x++) {
						if (binary[x*sized_character.rows + y] == '1') {
							horizontal_celled_projection[x*sized_character.rows + sy] = '1';
						}
					}
				}
			}

			if (SHOW_VERTICAL_CELLED_PROJECTION) {
				cout << "vertical celled projection" << endl << endl;
				for (int x = 0; x < sized_character.cols; x++) {
					for (int y = 0; y < sized_character.rows; y++) {
						cout << vertical_celled_projection[x*sized_character.rows + y];
					}
					cout << endl;
				}
				cout << endl;
			}


			if (SHOW_HORIZONTAL_CELLED_PROJECTION) {
				cout << "horizontal celled projection" << endl << endl;
				for (int x = 0; x < sized_character.cols; x++) {
					for (int y = 0; y < sized_character.rows; y++) {
						cout << horizontal_celled_projection[x*sized_character.rows + y];
					}
					cout << endl;
				}
				cout << endl;
			}


			if (WRITE_CHARACTER_IMAGE_FILE) {
				imwrite("output/character" + to_string(count) + ".jpg", sized_character);
				count++;
			}

			if (DESCRIBE_CHARACTER) {
				namedWindow("Janus", WINDOW_AUTOSIZE);
				imshow("Janus", sized_character);
				char c = waitKey(0);
				cout << c << endl;
			}

			if (0) cout << endl;

			tableau.push_back(character(sized_character, original_size, binary, vertical_celled_projection, horizontal_celled_projection, density));

			free(binary);
			free(ones);
			free(zeroes);
			free(density);
		}
		else {
			cout << "box out of bounds" << endl;
		}
		
	}

	cout << tableau.size() << " characters in tableau" << endl << endl;


	for (int x = 0; x < image.cols; x++) {

		for (int y = 0; y < image.rows; y++) {

			image.at<uchar>(y, x) = artifact[y*image.cols + x];

		}
	}


	free(artifact);
	free(characters);

	
	chrono::high_resolution_clock::time_point stop = chrono::high_resolution_clock::now();

	cout << "extracting: " << chrono::duration_cast<chrono::milliseconds>(stop - start).count() << " milliseconds" << endl;


	if (DISPLAY_AFTER_EXTRACTION) {
		
		imwrite("output/" + filename + ".jpg", image);

		namedWindow("Janus", WINDOW_AUTOSIZE);
		imshow("Janus", image);

		waitKey(0);

	}

	return tableau;
}